#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <assert.h>
#include <cstring> 
#include <vector>
#include <utility>
#include <cstdint> 

#define RED_ADD_THREADS 256

#define CHECK_CUDA(cmd) do { \
  hipError_t e = cmd; \
  if (e != hipSuccess) { \
    printf("CUDA error %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
    exit(EXIT_FAILURE); \
  } \
} while (0)

#define CHECK_NCCL(cmd) do { \
  ncclResult_t res = cmd; \
  if (res != ncclSuccess) { \
    printf("NCCL error %s:%d: '%s'\n", __FILE__, __LINE__, ncclGetErrorString(res)); \
    exit(EXIT_FAILURE); \
  } \
} while (0)

__global__ void reduce_add(float* dst, float* src, int count) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < count) {
    dst[i] += src[i];
  }
}

__global__ void gpu_sleep_kernel(clock_t sleep_cycles) {
  clock_t start = clock();
  while (clock() - start < sleep_cycles);
}

__global__ void fill_pattern(float* dst, float v, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (size_t i = idx; i < n; i += gridDim.x * blockDim.x)
    dst[i] = v;
}

clock_t calculate_sleep_cycles(float ms, int* devs) {
  hipSetDevice(devs[7]);

  // Query clock rate (in kHz)
  int clockRate_kHz;
  hipDeviceGetAttribute(&clockRate_kHz, hipDeviceAttributeClockRate, devs[3]);

  // Compute number of cycles to sleep
  clock_t sleep_cycles = static_cast<clock_t>(ms * clockRate_kHz);

  return sleep_cycles;
}

void direct_allreduce_helper(float** d_buffers, float** d_tempbufs, int* devs, hipStream_t* streams, ncclComm_t* comms, hipEvent_t start, hipEvent_t stop, int numRanks, size_t size) {
  for (int r = 0; r < numRanks; ++r) {
    hipSetDevice(devs[r]);
    hipStreamSynchronize(streams[r]);
  }

  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[0], size, ncclFloat, 7, comms[0], streams[0]);
  ncclRecv(d_tempbufs[0], size, ncclFloat, 7, comms[0], streams[0]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[7], size, ncclFloat, 0, comms[7], streams[7]);
  ncclRecv(d_tempbufs[7], size, ncclFloat, 0, comms[7], streams[7]);
  ncclGroupEnd();
  ncclGroupEnd();
  hipSetDevice(devs[0]);
  reduce_add<<<(size + 128 - 1) / 128, 128, 0, streams[0]>>>(d_buffers[0], d_tempbufs[0], size);
  hipSetDevice(devs[7]);
  reduce_add<<<(size + 128 - 1) / 128, 128, 0, streams[7]>>>(d_buffers[7], d_tempbufs[7], size);

  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[0], size, ncclFloat, 1, comms[0], streams[0]);
  ncclRecv(d_buffers[1], size, ncclFloat, 0, comms[1], streams[1]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[7], size, ncclFloat, 6, comms[7], streams[7]);
  ncclRecv(d_buffers[6], size, ncclFloat, 7, comms[6], streams[6]);
  ncclGroupEnd();
  ncclGroupEnd();

  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[0], size, ncclFloat, 2, comms[0], streams[0]);
  ncclRecv(d_buffers[2], size, ncclFloat, 0, comms[2], streams[2]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[1], size, ncclFloat, 3, comms[1], streams[1]);
  ncclRecv(d_buffers[3], size, ncclFloat, 1, comms[3], streams[3]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[6], size, ncclFloat, 4, comms[6], streams[6]);
  ncclRecv(d_buffers[4], size, ncclFloat, 6, comms[4], streams[4]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[7], size, ncclFloat, 5, comms[7], streams[7]);
  ncclRecv(d_buffers[5], size, ncclFloat, 7, comms[5], streams[5]);
  ncclGroupEnd();
  ncclGroupEnd();
  hipSetDevice(devs[0]);
  hipEventRecord(stop, streams[0]);
  hipEventSynchronize(stop);
}

void direct_allreduce_delay(float** d_buffers, float** d_tempbufs, int* devs, hipStream_t* streams, ncclComm_t* comms, ncclComm_t* subComms, hipEvent_t start, hipEvent_t stop, int numRanks, size_t size, clock_t sleep_cycles) {
  hipSetDevice(devs[0]);
  hipEventRecord(start, streams[0]);

  // sleep the straggler
  hipSetDevice(devs[7]);
  gpu_sleep_kernel<<<1, 1, 0, streams[7]>>>(sleep_cycles);

  // Synchronize to make sure everything is idle
  for (int r = 0; r < numRanks - 1; ++r) {
    hipSetDevice(devs[r]);
    hipStreamSynchronize(streams[r]);
  }

  ncclGroupStart();
  for (int r = 0; r < numRanks - 1; ++r) {
    hipSetDevice(devs[r]);
    ncclAllReduce(d_buffers[r], d_buffers[r], size, ncclFloat, ncclSum, subComms[r], streams[r]);
  }
  ncclGroupEnd();

  direct_allreduce_helper(d_buffers, d_tempbufs, devs, streams, comms, start, stop, numRanks, size);
}

void direct_allreduce(float** d_buffers, float** d_tempbufs, int* devs, hipStream_t* streams, ncclComm_t* comms, hipEvent_t start, hipEvent_t stop, int numRanks, size_t size) {
  hipSetDevice(devs[0]);
  hipEventRecord(start, streams[0]);
  
  direct_allreduce_helper(d_buffers, d_tempbufs, devs, streams, comms, start, stop, numRanks, size);
}

void ring_allreduce_helper(float** d_buffers, float** d_tempbufs, int* devs, hipStream_t* streams, ncclComm_t* comms, hipEvent_t start, hipEvent_t stop, int numRanks, size_t chunkSize) {

  // Make sure everything is initialized
  for (int i = 0; i < numRanks; ++i) {
    CHECK_CUDA(hipSetDevice(devs[i]));
    CHECK_CUDA(hipStreamSynchronize(streams[i]));  // ensure idle
  }
    // Ring Reduce-Scatter
    for (int step = 1; step < numRanks; ++step) {
      ncclGroupStart();
      for (int r = 0; r < numRanks; ++r) {
        int sendTo = (r + 1) % numRanks;
        int recvFrom = (r - 1 + numRanks) % numRanks;
        int sendChunk = (r - step + numRanks) % numRanks;
        int recvChunk = (r - step - 1 + numRanks) % numRanks;

        float* sendPtr = d_buffers[r] + sendChunk * chunkSize;
        float* recvPtr = d_tempbufs[r];
        ncclGroupStart();
        ncclSend(sendPtr, chunkSize, ncclFloat, sendTo, comms[r], streams[r]);
        ncclRecv(recvPtr, chunkSize, ncclFloat, recvFrom, comms[r], streams[r]);
        ncclGroupEnd();
      }
      ncclGroupEnd();
      for (int r = 0; r < numRanks; ++r) {
        hipSetDevice(devs[r]);
        int recvChunk = (r - step - 1 + numRanks) % numRanks;
        reduce_add<<<(chunkSize + RED_ADD_THREADS - 1) / RED_ADD_THREADS, RED_ADD_THREADS, 0, streams[r]>>>(d_buffers[r] + recvChunk * chunkSize, d_tempbufs[r], chunkSize);
      }
    }

    for (int step = 0; step < numRanks - 1; ++step) {
      ncclGroupStart();
      for (int r = 0; r < numRanks; ++r) {
        int sendTo = (r + 1) % numRanks;
        int recvFrom = (r - 1 + numRanks) % numRanks;
        int sendChunk = (r - step + numRanks) % numRanks;
        int recvChunk = (r - step - 1 + numRanks) % numRanks;

        float* sendPtr = d_buffers[r] + sendChunk * chunkSize;
        float* recvPtr = d_buffers[r] + recvChunk * chunkSize;
        ncclGroupStart();
        ncclSend(sendPtr, chunkSize, ncclFloat, sendTo, comms[r], streams[r]);
        ncclRecv(recvPtr, chunkSize, ncclFloat, recvFrom, comms[r], streams[r]);
        ncclGroupEnd();
      }
      ncclGroupEnd();
    }

    hipSetDevice(devs[0]);
    hipEventRecord(stop, streams[0]);
    hipEventSynchronize(stop);
}


void ring_allreduce_delay(float** d_buffers, float** d_tempbufs, int* devs, hipStream_t* streams, ncclComm_t* comms, hipEvent_t start, hipEvent_t stop, int numRanks, size_t size, clock_t sleep_cycles) {
  size_t chunkSize = size / numRanks;

  hipSetDevice(devs[0]);
  hipEventRecord(start, streams[0]);
  hipSetDevice(devs[7]);
  gpu_sleep_kernel<<<1, 1, 0, streams[7]>>>(sleep_cycles);

  ring_allreduce_helper(d_buffers, d_tempbufs, devs, streams, comms, start, stop, numRanks, chunkSize);
}

void ring_allreduce(float** d_buffers, float** d_tempbufs, int* devs, hipStream_t* streams, ncclComm_t* comms, hipEvent_t start, hipEvent_t stop, int numRanks, size_t size) {
  size_t chunkSize = size / numRanks;

  hipSetDevice(devs[0]);
  hipEventRecord(start, streams[0]);
  ring_allreduce_helper(d_buffers, d_tempbufs, devs, streams, comms, start, stop, numRanks, chunkSize);
}


void rhd_allreduce_helper(float** d_buffers, float** d_tempbufs, int* devs, hipStream_t* streams, ncclComm_t* comms, hipEvent_t start, hipEvent_t stop, int numRanks, size_t chunkSize) {
  // Synchronize to make sure everything is idle
  for (int r = 0; r < numRanks; ++r) {
    hipSetDevice(devs[r]);
    hipStreamSynchronize(streams[r]);
  }

  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[0] + (4 * chunkSize), chunkSize * 4, ncclFloat, 1, comms[0], streams[0]);
  ncclRecv(d_tempbufs[0], chunkSize * 4, ncclFloat, 1, comms[0], streams[0]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[1], chunkSize * 4, ncclFloat, 0, comms[1], streams[1]);
  ncclRecv(d_tempbufs[1], chunkSize * 4, ncclFloat, 0, comms[1], streams[1]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[2] + (4 * chunkSize), chunkSize * 4, ncclFloat, 3, comms[2], streams[2]);
  ncclRecv(d_tempbufs[2], chunkSize * 4, ncclFloat, 3, comms[2], streams[2]);
  ncclGroupEnd();
    
  ncclGroupStart();
  ncclSend(d_buffers[3], chunkSize * 4, ncclFloat, 2, comms[3], streams[3]);
  ncclRecv(d_tempbufs[3], chunkSize * 4, ncclFloat, 2, comms[3], streams[3]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[4] + (4 * chunkSize), chunkSize * 4, ncclFloat, 5, comms[4], streams[4]);
  ncclRecv(d_tempbufs[4], chunkSize * 4, ncclFloat, 5, comms[4], streams[4]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[5], chunkSize * 4, ncclFloat, 4, comms[5], streams[5]);
  ncclRecv(d_tempbufs[5], chunkSize * 4, ncclFloat, 4, comms[5], streams[5]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[6] + (4 * chunkSize), chunkSize * 4, ncclFloat, 7, comms[6], streams[6]);
  ncclRecv(d_tempbufs[6], chunkSize * 4, ncclFloat, 7, comms[6], streams[6]);
  ncclGroupEnd();
    
  ncclGroupStart();
  ncclSend(d_buffers[7], chunkSize * 4, ncclFloat, 6, comms[7], streams[7]);
  ncclRecv(d_tempbufs[7], chunkSize * 4, ncclFloat, 6, comms[7], streams[7]);
  ncclGroupEnd();
  ncclGroupEnd();
  
  for (int r = 0; r < numRanks; ++r) {
    hipSetDevice(devs[r]);
    int recvChunk = r % 2;
    float* dstPtr = d_buffers[r] + (4 * recvChunk * chunkSize);
    int numBlocks = (4 * chunkSize + RED_ADD_THREADS - 1) / RED_ADD_THREADS;
    reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[r]>>>(dstPtr, d_tempbufs[r], 4 * chunkSize);
  }

  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[0] + (2 * chunkSize), 2 * chunkSize, ncclFloat, 2, comms[0], streams[0]);
  ncclRecv(d_tempbufs[0], 2 * chunkSize, ncclFloat, 2, comms[0], streams[0]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[1] + (6 * chunkSize), 2 * chunkSize, ncclFloat, 3, comms[1], streams[1]);
  ncclRecv(d_tempbufs[1], 2 * chunkSize, ncclFloat, 3, comms[1], streams[1]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[2], 2 * chunkSize, ncclFloat, 0, comms[2], streams[2]);
  ncclRecv(d_tempbufs[2], 2 * chunkSize, ncclFloat, 0, comms[2], streams[2]);
  ncclGroupEnd();
    
  ncclGroupStart();
  ncclSend(d_buffers[3] + (4 * chunkSize), 2 * chunkSize, ncclFloat, 1, comms[3], streams[3]);
  ncclRecv(d_tempbufs[3], 2 * chunkSize, ncclFloat, 1, comms[3], streams[3]);
  ncclGroupEnd();

  ncclGroupStart(); 
  ncclSend(d_buffers[4] + (2 * chunkSize), 2 * chunkSize, ncclFloat, 6, comms[4], streams[4]);
  ncclRecv(d_tempbufs[4], 2 * chunkSize, ncclFloat, 6, comms[4], streams[4]);
  ncclGroupEnd();
  
  ncclGroupStart();
  ncclSend(d_buffers[5] + (6 * chunkSize), 2 * chunkSize, ncclFloat, 7, comms[5], streams[5]);
  ncclRecv(d_tempbufs[5], 2 * chunkSize, ncclFloat, 7, comms[5], streams[5]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[6], 2 * chunkSize, ncclFloat, 4, comms[6], streams[6]);
  ncclRecv(d_tempbufs[6], 2 * chunkSize, ncclFloat, 4, comms[6], streams[6]);
  ncclGroupEnd();
    
  ncclGroupStart();
  ncclSend(d_buffers[7] + (4 * chunkSize), 2 * chunkSize, ncclFloat, 5, comms[7], streams[7]);
  ncclRecv(d_tempbufs[7], 2 * chunkSize, ncclFloat, 5, comms[7], streams[7]);
  ncclGroupEnd();
  ncclGroupEnd();

  int numBlocks = (2 * chunkSize + RED_ADD_THREADS - 1) / RED_ADD_THREADS;

  hipSetDevice(devs[0]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[0]>>>(d_buffers[0], d_tempbufs[0], 2 * chunkSize);
  hipSetDevice(devs[2]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[2]>>>(d_buffers[2] + (2 * chunkSize), d_tempbufs[2], 2 * chunkSize);
  hipSetDevice(devs[1]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[1]>>>(d_buffers[1] + (4 * chunkSize), d_tempbufs[1], 2 * chunkSize);
  hipSetDevice(devs[3]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[3]>>>(d_buffers[3] + (6 * chunkSize), d_tempbufs[3], 2 * chunkSize);

  hipSetDevice(devs[4]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[4]>>>(d_buffers[4], d_tempbufs[4], 2 * chunkSize);
  hipSetDevice(devs[6]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[6]>>>(d_buffers[6] + (2 * chunkSize), d_tempbufs[6], 2 * chunkSize);
  hipSetDevice(devs[5]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[5]>>>(d_buffers[5] + (4 * chunkSize), d_tempbufs[5], 2 * chunkSize);
  hipSetDevice(devs[7]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[7]>>>(d_buffers[7] + (6 * chunkSize), d_tempbufs[7], 2 * chunkSize);

  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[0] + chunkSize, chunkSize, ncclFloat, 4, comms[0], streams[0]);
  ncclRecv(d_tempbufs[0], chunkSize, ncclFloat, 4, comms[0], streams[0]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[1] + (5 * chunkSize), chunkSize, ncclFloat, 5, comms[1], streams[1]);
  ncclRecv(d_tempbufs[1], chunkSize, ncclFloat, 5, comms[1], streams[1]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[2] + (3 * chunkSize), chunkSize, ncclFloat, 6, comms[2], streams[2]);
  ncclRecv(d_tempbufs[2], chunkSize, ncclFloat, 6, comms[2], streams[2]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[3] + (7 * chunkSize), chunkSize, ncclFloat, 7, comms[3], streams[3]);
  ncclRecv(d_tempbufs[3], chunkSize, ncclFloat, 7, comms[3], streams[3]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[4], chunkSize, ncclFloat, 0, comms[4], streams[4]);
  ncclRecv(d_tempbufs[4], chunkSize, ncclFloat, 0, comms[4], streams[4]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[5] + (4 * chunkSize), chunkSize, ncclFloat, 1, comms[5], streams[5]);
  ncclRecv(d_tempbufs[5], chunkSize, ncclFloat, 1, comms[5], streams[5]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[6] + (2 * chunkSize), chunkSize, ncclFloat, 2, comms[6], streams[6]);
  ncclRecv(d_tempbufs[6], chunkSize, ncclFloat, 2, comms[6], streams[6]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[7] + (6 * chunkSize), chunkSize, ncclFloat, 3, comms[7], streams[7]);
  ncclRecv(d_tempbufs[7], chunkSize, ncclFloat, 3, comms[7], streams[7]);
  ncclGroupEnd();
  ncclGroupEnd();

  numBlocks = (chunkSize + RED_ADD_THREADS - 1) / RED_ADD_THREADS;
  hipSetDevice(devs[0]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[0]>>>(d_buffers[0], d_tempbufs[0], chunkSize);
  hipSetDevice(devs[4]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[4]>>>(d_buffers[4] + chunkSize, d_tempbufs[4], chunkSize);

  hipSetDevice(devs[2]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[2]>>>(d_buffers[2] + (2 * chunkSize), d_tempbufs[2], chunkSize);
  hipSetDevice(devs[6]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[6]>>>(d_buffers[6] + (3 * chunkSize), d_tempbufs[6], chunkSize);

  hipSetDevice(devs[1]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[1]>>>(d_buffers[1] + (4 * chunkSize), d_tempbufs[1], chunkSize);
  hipSetDevice(devs[5]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[5]>>>(d_buffers[5] + (5 * chunkSize), d_tempbufs[5], chunkSize);

  hipSetDevice(devs[3]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[3]>>>(d_buffers[3] + (6 * chunkSize), d_tempbufs[3], chunkSize);
  hipSetDevice(devs[7]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[7]>>>(d_buffers[7] + (7 * chunkSize), d_tempbufs[7], chunkSize);

  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[0], chunkSize, ncclFloat, 4, comms[0], streams[0]);
  ncclRecv(d_buffers[0] + chunkSize, chunkSize, ncclFloat, 4, comms[0], streams[0]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[1] + (4 * chunkSize), chunkSize, ncclFloat, 5, comms[1], streams[1]);
  ncclRecv(d_buffers[1] + (5 * chunkSize), chunkSize, ncclFloat, 5, comms[1], streams[1]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[2] + (2 * chunkSize), chunkSize, ncclFloat, 6, comms[2], streams[2]);
  ncclRecv(d_buffers[2] + (3 * chunkSize), chunkSize, ncclFloat, 6, comms[2], streams[2]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[3] + (6 * chunkSize), chunkSize, ncclFloat, 7, comms[3], streams[3]);
  ncclRecv(d_buffers[3] + (7 * chunkSize), chunkSize, ncclFloat, 7, comms[3], streams[3]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[4] + chunkSize, chunkSize, ncclFloat, 0, comms[4], streams[4]);
  ncclRecv(d_buffers[4], chunkSize, ncclFloat, 0, comms[4], streams[4]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[5] + (5 * chunkSize), chunkSize, ncclFloat, 1, comms[5], streams[5]);
  ncclRecv(d_buffers[5] + (4 * chunkSize), chunkSize, ncclFloat, 1, comms[5], streams[5]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[6] + (3 * chunkSize), chunkSize, ncclFloat, 2, comms[6], streams[6]);
  ncclRecv(d_buffers[6] + (2 * chunkSize), chunkSize, ncclFloat, 2, comms[6], streams[6]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[7] + (7 * chunkSize), chunkSize, ncclFloat, 3, comms[7], streams[7]);
  ncclRecv(d_buffers[7] + (6 * chunkSize), chunkSize, ncclFloat, 3, comms[7], streams[7]);
  ncclGroupEnd();
  ncclGroupEnd();

  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[0], 2 * chunkSize, ncclFloat, 2, comms[0], streams[0]);
  ncclRecv(d_buffers[0] + (2 * chunkSize), 2 * chunkSize, ncclFloat, 2, comms[0], streams[0]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[1] + (4 * chunkSize), 2 * chunkSize, ncclFloat, 3, comms[1], streams[1]);
  ncclRecv(d_buffers[1] + (6 * chunkSize), 2 * chunkSize, ncclFloat, 3, comms[1], streams[1]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[2]  + (2 * chunkSize), 2 * chunkSize, ncclFloat, 0, comms[2], streams[2]);
  ncclRecv(d_buffers[2], 2 * chunkSize, ncclFloat, 0, comms[2], streams[2]);
  ncclGroupEnd();
    
  ncclGroupStart();
  ncclSend(d_buffers[3] + (6 * chunkSize), 2 * chunkSize, ncclFloat, 1, comms[3], streams[3]);
  ncclRecv(d_buffers[3] + (4 * chunkSize), 2 * chunkSize, ncclFloat, 1, comms[3], streams[3]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[4], 2 * chunkSize, ncclFloat, 6, comms[4], streams[4]);
  ncclRecv(d_buffers[4] + (2 * chunkSize), 2 * chunkSize, ncclFloat, 6, comms[4], streams[4]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[5] + (4 * chunkSize), 2 * chunkSize, ncclFloat, 7, comms[5], streams[5]);
  ncclRecv(d_buffers[5] + (6 * chunkSize), 2 * chunkSize, ncclFloat, 7, comms[5], streams[5]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[6] + (2 * chunkSize), 2 * chunkSize, ncclFloat, 4, comms[6], streams[6]);
  ncclRecv(d_buffers[6], 2 * chunkSize, ncclFloat, 4, comms[6], streams[6]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[7] + (6 * chunkSize), 2 * chunkSize, ncclFloat, 5, comms[7], streams[7]);
  ncclRecv(d_buffers[7] + (4 * chunkSize), 2 * chunkSize, ncclFloat, 5, comms[7], streams[7]);
  ncclGroupEnd();
  ncclGroupEnd();

  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[0], chunkSize * 4, ncclFloat, 1, comms[0], streams[0]);
  ncclRecv(d_buffers[0] + (4 * chunkSize), chunkSize * 4, ncclFloat, 1, comms[0], streams[0]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[1] + (4 * chunkSize), chunkSize * 4, ncclFloat, 0, comms[1], streams[1]);
  ncclRecv(d_buffers[1], chunkSize * 4, ncclFloat, 0, comms[1], streams[1]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[2], chunkSize * 4, ncclFloat, 3, comms[2], streams[2]);
  ncclRecv(d_buffers[2] + (4 * chunkSize), chunkSize * 4, ncclFloat, 3, comms[2], streams[2]);
  ncclGroupEnd();
  
  ncclGroupStart();
  ncclSend(d_buffers[3] + (4 * chunkSize), chunkSize * 4, ncclFloat, 2, comms[3], streams[3]);
  ncclRecv(d_buffers[3], chunkSize * 4, ncclFloat, 2, comms[3], streams[3]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[4], chunkSize * 4, ncclFloat, 5, comms[4], streams[4]);
  ncclRecv(d_buffers[4] + (4 * chunkSize), chunkSize * 4, ncclFloat, 5, comms[4], streams[4]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[5] + (4 * chunkSize), chunkSize * 4, ncclFloat, 4, comms[5], streams[5]);
  ncclRecv(d_buffers[5], chunkSize * 4, ncclFloat, 4, comms[5], streams[5]);
  ncclGroupEnd();

  ncclGroupStart();
  ncclSend(d_buffers[6], chunkSize * 4, ncclFloat, 7, comms[6], streams[6]);
  ncclRecv(d_buffers[6] + (4 * chunkSize), chunkSize * 4, ncclFloat, 7, comms[6], streams[6]);
  ncclGroupEnd();
    
  ncclGroupStart();
  ncclSend(d_buffers[7] + (4 * chunkSize), chunkSize * 4, ncclFloat, 6, comms[7], streams[7]);
  ncclRecv(d_buffers[7], chunkSize * 4, ncclFloat, 6, comms[7], streams[7]);
  ncclGroupEnd();
  ncclGroupEnd();

  hipSetDevice(devs[0]);
  hipEventRecord(stop, streams[0]);
  hipEventSynchronize(stop);
}


void rhd_allreduce_delay(float** d_buffers, float** d_tempbufs, int* devs, hipStream_t* streams, ncclComm_t* comms, hipEvent_t start, hipEvent_t stop, int numRanks, size_t size, clock_t sleep_cycles) {
  int chunkSize = size / numRanks;
  hipSetDevice(devs[0]);
  hipEventRecord(start, streams[0]);

  // sleep the straggler
  hipSetDevice(devs[7]);
  gpu_sleep_kernel<<<1, 1, 0, streams[7]>>>(sleep_cycles);

  rhd_allreduce_helper(d_buffers, d_tempbufs, devs, streams, comms, start, stop, numRanks, chunkSize);
}

void rhd_allreduce(float** d_buffers, float** d_tempbufs, int* devs, hipStream_t* streams, ncclComm_t* comms, hipEvent_t start, hipEvent_t stop, int numRanks, size_t size) {
  int chunkSize = size / numRanks;
  hipSetDevice(devs[0]);
  hipEventRecord(start, streams[0]);
  rhd_allreduce_helper(d_buffers, d_tempbufs, devs, streams, comms, start, stop, numRanks, chunkSize);
}


void stragglar_allreduce_helper(float** d_buffers, float** d_tempbufs, int* devs, hipStream_t* streams, ncclComm_t* comms, hipEvent_t start, hipEvent_t stop, int numRanks, size_t chunkSize) {
  int numBlocks = (chunkSize + RED_ADD_THREADS - 1) / RED_ADD_THREADS;

  // Synchronize to make sure everything is idle
  for (int r = 0; r < numRanks; ++r) {
    hipSetDevice(devs[r]);
    hipStreamSynchronize(streams[r]);
  }

  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[0], chunkSize, ncclFloat, 7, comms[0], streams[0]);
  ncclRecv(d_tempbufs[0], chunkSize, ncclFloat, 7, comms[0], streams[0]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[7], chunkSize, ncclFloat, 0, comms[7], streams[7]);
  ncclRecv(d_tempbufs[7], chunkSize, ncclFloat, 0, comms[7], streams[7]);
  ncclGroupEnd();
  ncclGroupEnd();
  hipSetDevice(devs[0]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[0]>>>(d_buffers[0], d_tempbufs[0], chunkSize);
  
  hipSetDevice(devs[7]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[7]>>>(d_buffers[7], d_tempbufs[7], chunkSize);

  // step 2
  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[1] + chunkSize, chunkSize, ncclFloat, 7, comms[1], streams[1]);
  ncclRecv(d_tempbufs[1], chunkSize, ncclFloat, 7, comms[1], streams[1]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[7] + chunkSize, chunkSize, ncclFloat, 1, comms[7], streams[7]);
  ncclRecv(d_tempbufs[7], chunkSize, ncclFloat, 1, comms[7], streams[7]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[0], chunkSize, ncclFloat, 3, comms[0], streams[0]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[3], chunkSize, ncclFloat, 0, comms[3], streams[3]);
  ncclGroupEnd();
  ncclGroupEnd();
 
  hipSetDevice(devs[1]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[1]>>>(d_buffers[1] + chunkSize, d_tempbufs[1], chunkSize);
  hipSetDevice(devs[7]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[7]>>>(d_buffers[7] + chunkSize, d_tempbufs[7], chunkSize);

  // step 3
  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[7] + 2 * chunkSize, chunkSize, ncclFloat, 2, comms[7], streams[7]);
  ncclRecv(d_tempbufs[7], chunkSize, ncclFloat, 2, comms[7], streams[7]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[2] + 2 * chunkSize, chunkSize, ncclFloat, 7, comms[2], streams[2]);
  ncclRecv(d_tempbufs[2], chunkSize, ncclFloat, 7, comms[2], streams[2]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[1] + chunkSize, chunkSize, ncclFloat, 4, comms[1], streams[1]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[4] + chunkSize, chunkSize, ncclFloat, 1, comms[4], streams[4]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[0], chunkSize, ncclFloat, 5, comms[0], streams[0]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[5], chunkSize, ncclFloat, 0, comms[5], streams[5]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[3], chunkSize, ncclFloat, 6, comms[3], streams[3]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[6], chunkSize, ncclFloat, 3, comms[6], streams[6]);
  ncclGroupEnd();
  ncclGroupEnd();
  
  hipSetDevice(devs[2]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[2]>>>(d_buffers[2] + 2 * chunkSize, d_tempbufs[2], chunkSize);
  hipSetDevice(devs[7]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[7]>>>( d_buffers[7] + 2 * chunkSize, d_tempbufs[7], chunkSize);

  // step 4
  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[7] + 3 * chunkSize, chunkSize, ncclFloat, 3, comms[7], streams[7]);
  ncclRecv(d_tempbufs[7], chunkSize, ncclFloat, 3, comms[7], streams[7]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[3] + 3 * chunkSize, chunkSize, ncclFloat, 7, comms[3], streams[3]);
  ncclRecv(d_tempbufs[3], chunkSize, ncclFloat, 7, comms[3], streams[3]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[0], chunkSize, ncclFloat, 2, comms[0], streams[0]);
  ncclRecv(d_buffers[0] + 2 * chunkSize, chunkSize, ncclFloat, 2, comms[0], streams[0]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[2] + (2 * chunkSize), chunkSize, ncclFloat, 0, comms[2], streams[2]);
  ncclRecv(d_buffers[2], chunkSize, ncclFloat, 0, comms[2], streams[2]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[1] + chunkSize, chunkSize, ncclFloat, 5, comms[1], streams[1]);
  ncclRecv(d_buffers[1], chunkSize, ncclFloat, 5, comms[1], streams[1]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[5], chunkSize, ncclFloat, 1, comms[5], streams[5]);
  ncclRecv(d_buffers[5] + chunkSize, chunkSize, ncclFloat, 1, comms[5], streams[5]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[4] + chunkSize, chunkSize, ncclFloat, 6, comms[4], streams[4]);
  ncclRecv(d_buffers[4], chunkSize, ncclFloat, 6, comms[4], streams[4]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[6], chunkSize, ncclFloat, 4, comms[6], streams[6]);
  ncclRecv(d_buffers[6] + chunkSize, chunkSize, ncclFloat, 4, comms[6], streams[6]);
  ncclGroupEnd();
  ncclGroupEnd();
  hipSetDevice(devs[3]);
  hipStreamSynchronize(streams[3]);
  hipStreamSynchronize(streams[7]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[3]>>>(d_buffers[3] + 3 * chunkSize, d_tempbufs[3], chunkSize);
  hipSetDevice(devs[7]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[7]>>>( d_buffers[7] + 3 * chunkSize, d_tempbufs[7], chunkSize);

  // step 5
  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[7] + 4 * chunkSize, chunkSize, ncclFloat, 4, comms[7], streams[7]);
  ncclRecv(d_tempbufs[7], chunkSize, ncclFloat, 4, comms[7], streams[7]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[4] + 4 * chunkSize, chunkSize, ncclFloat, 7, comms[4], streams[4]);
  ncclRecv(d_tempbufs[4], chunkSize, ncclFloat, 7, comms[4], streams[4]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[3] + 3 * chunkSize, chunkSize, ncclFloat, 1, comms[3], streams[3]);
  ncclRecv(d_buffers[3] + chunkSize, chunkSize, ncclFloat, 1, comms[3], streams[3]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[1] + chunkSize, chunkSize, ncclFloat, 3, comms[1], streams[1]);
  ncclRecv(d_buffers[1] + 3 * chunkSize, chunkSize, ncclFloat, 3, comms[1], streams[1]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[0] + 2 * chunkSize, chunkSize, ncclFloat, 5, comms[0], streams[0]);
  ncclRecv(d_buffers[0] + chunkSize, chunkSize, ncclFloat, 5, comms[0], streams[0]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[5] + chunkSize, chunkSize, ncclFloat, 0, comms[5], streams[5]);
  ncclRecv(d_buffers[5] + 2 * chunkSize, chunkSize, ncclFloat, 0, comms[5], streams[5]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[2] + 2 * chunkSize, chunkSize, ncclFloat, 6, comms[2], streams[2]);
  ncclRecv(d_buffers[2] + chunkSize, chunkSize, ncclFloat, 6, comms[2], streams[2]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[6] + chunkSize, chunkSize, ncclFloat, 2, comms[6], streams[6]);
  ncclRecv(d_buffers[6] + 2 * chunkSize, chunkSize, ncclFloat, 2, comms[6], streams[6]);
  ncclGroupEnd();
  ncclGroupEnd();
  hipSetDevice(devs[4]);
  hipStreamSynchronize(streams[4]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[4]>>>(d_buffers[4] + 4 * chunkSize, d_tempbufs[4], chunkSize);
  hipSetDevice(devs[7]);
  hipStreamSynchronize(streams[7]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[7]>>>( d_buffers[7] + 4 * chunkSize, d_tempbufs[7], chunkSize);

  // step 6
  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[7] + 5 * chunkSize, chunkSize, ncclFloat, 5, comms[7], streams[7]);
  ncclRecv(d_tempbufs[7], chunkSize, ncclFloat, 5, comms[7], streams[7]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_tempbufs[5], chunkSize, ncclFloat, 7, comms[5], streams[5]);
  ncclSend(d_buffers[5] + 5 * chunkSize, chunkSize, ncclFloat, 7, comms[5], streams[5]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[4] + 4 * chunkSize, chunkSize, ncclFloat, 2, comms[4], streams[4]);
  ncclRecv(d_buffers[4] + 2 * chunkSize, chunkSize, ncclFloat, 2, comms[4], streams[4]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[2] + 4 * chunkSize, chunkSize, ncclFloat, 4, comms[2], streams[2]);
  ncclSend(d_buffers[2] + 2 * chunkSize, chunkSize, ncclFloat, 4, comms[2], streams[2]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[0] + 2 * chunkSize, chunkSize, ncclFloat, 3, comms[0], streams[0]);
  ncclRecv(d_buffers[0] + 3 * chunkSize, chunkSize, ncclFloat, 3, comms[0], streams[0]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[3] + 2 * chunkSize, chunkSize, ncclFloat, 0, comms[3], streams[3]);
  ncclSend(d_buffers[3] + 3 * chunkSize, chunkSize, ncclFloat, 0, comms[3], streams[3]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[1] + 3 * chunkSize, chunkSize, ncclFloat, 6, comms[1], streams[1]);
  ncclRecv(d_buffers[1] + 2 * chunkSize, chunkSize, ncclFloat, 6, comms[1], streams[1]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[6] + 3 * chunkSize, chunkSize, ncclFloat, 1, comms[6], streams[6]);
  ncclSend(d_buffers[6] + 2 * chunkSize, chunkSize, ncclFloat, 1, comms[6], streams[6]);
  ncclGroupEnd();
  ncclGroupEnd();

  hipSetDevice(devs[5]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[5]>>>(d_buffers[5] + 5 * chunkSize, d_tempbufs[5], chunkSize);
  hipSetDevice(devs[7]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[7]>>>( d_buffers[7] + 5 * chunkSize, d_tempbufs[7], chunkSize);

  // step 7
  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[7] + 6 * chunkSize, chunkSize, ncclFloat, 6, comms[7], streams[7]);
  ncclRecv(d_tempbufs[7], chunkSize, ncclFloat, 6, comms[7], streams[7]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_tempbufs[6], chunkSize, ncclFloat, 7, comms[6], streams[6]);
  ncclSend(d_buffers[6] + 6 * chunkSize, chunkSize, ncclFloat, 7, comms[6], streams[6]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[5] + 5 * chunkSize, chunkSize, ncclFloat, 3, comms[5], streams[5]);
  ncclRecv(d_buffers[5] + 3 * chunkSize, chunkSize, ncclFloat, 3, comms[5], streams[5]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[3] + 5 * chunkSize, chunkSize, ncclFloat, 5, comms[3], streams[3]);
  ncclSend(d_buffers[3] + 3 * chunkSize, chunkSize, ncclFloat, 5, comms[3], streams[3]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[0] + 3 * chunkSize, chunkSize, ncclFloat, 2, comms[0], streams[0]);
  ncclRecv(d_buffers[0] + 4 * chunkSize, chunkSize, ncclFloat, 2, comms[0], streams[0]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[2] + 3 * chunkSize, chunkSize, ncclFloat, 0, comms[2], streams[2]);
  ncclSend(d_buffers[2] + 4 * chunkSize, chunkSize, ncclFloat, 0, comms[2], streams[2]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[1] + 3 * chunkSize, chunkSize, ncclFloat, 4, comms[1], streams[1]);
  ncclRecv(d_buffers[1] + 4 * chunkSize, chunkSize, ncclFloat, 4, comms[1], streams[1]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[4] + 3 * chunkSize, chunkSize, ncclFloat, 1, comms[4], streams[4]);
  ncclSend(d_buffers[4] + 4 * chunkSize, chunkSize, ncclFloat, 1, comms[4], streams[4]);
  ncclGroupEnd();
  ncclGroupEnd();
  hipSetDevice(devs[6]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[6]>>>(d_buffers[6] + 6 * chunkSize, d_tempbufs[6], chunkSize);
  hipSetDevice(devs[7]);
  reduce_add<<<numBlocks, RED_ADD_THREADS, 0, streams[7]>>>( d_buffers[7] + 6 * chunkSize, d_tempbufs[7], chunkSize);

  // step 8
  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[6] + 6 * chunkSize, chunkSize, ncclFloat, 4, comms[6], streams[6]);
  ncclRecv(d_buffers[6] + 4 * chunkSize, chunkSize, ncclFloat, 4, comms[6], streams[6]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[4] + 6 * chunkSize, chunkSize, ncclFloat, 6, comms[4], streams[4]);
  ncclSend(d_buffers[4] + 4 * chunkSize, chunkSize, ncclFloat, 6, comms[4], streams[4]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[0] + 4 * chunkSize, chunkSize, ncclFloat, 3, comms[0], streams[0]);
  ncclRecv(d_buffers[0] + 5 * chunkSize, chunkSize, ncclFloat, 3, comms[0], streams[0]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[3] + 4 * chunkSize, chunkSize, ncclFloat, 0, comms[3], streams[3]);
  ncclSend(d_buffers[3] + 5 * chunkSize, chunkSize, ncclFloat, 0, comms[3], streams[3]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[1] + 4 * chunkSize, chunkSize, ncclFloat, 5, comms[1], streams[1]);
  ncclRecv(d_buffers[1] + 5 * chunkSize, chunkSize, ncclFloat, 5, comms[1], streams[1]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[5] + 4 * chunkSize, chunkSize, ncclFloat, 1, comms[5], streams[5]);
  ncclSend(d_buffers[5] + 5 * chunkSize, chunkSize, ncclFloat, 1, comms[5], streams[5]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[7] + 6 * chunkSize, chunkSize, ncclFloat, 2, comms[7], streams[7]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[2] + 6 * chunkSize, chunkSize, ncclFloat, 7, comms[2], streams[2]);
  ncclGroupEnd();
  ncclGroupEnd();

  // step 9
  ncclGroupStart();
  ncclGroupStart();
  ncclSend(d_buffers[0] + 5 * chunkSize, chunkSize, ncclFloat, 2, comms[0], streams[0]);
  ncclRecv(d_buffers[0] + 6 * chunkSize, chunkSize, ncclFloat, 2, comms[0], streams[0]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[2] + 5 * chunkSize, chunkSize, ncclFloat, 0, comms[2], streams[2]);
  ncclSend(d_buffers[2] + 6 * chunkSize, chunkSize, ncclFloat, 0, comms[2], streams[2]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[1] + 5 * chunkSize, chunkSize, ncclFloat, 4, comms[1], streams[1]);
  ncclRecv(d_buffers[1] + 6 * chunkSize, chunkSize, ncclFloat, 4, comms[1], streams[1]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[4] + 5 * chunkSize, chunkSize, ncclFloat, 1, comms[4], streams[4]);
  ncclSend(d_buffers[4] + 6 * chunkSize, chunkSize, ncclFloat, 1, comms[4], streams[4]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[3] + 5 * chunkSize, chunkSize, ncclFloat, 6, comms[3], streams[3]);
  ncclRecv(d_buffers[3] + 6 * chunkSize, chunkSize, ncclFloat, 6, comms[3], streams[3]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[6] + 5 * chunkSize, chunkSize, ncclFloat, 3, comms[6], streams[6]);
  ncclSend(d_buffers[6] + 6 * chunkSize, chunkSize, ncclFloat, 3, comms[6], streams[6]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclSend(d_buffers[7] + 6 * chunkSize, chunkSize, ncclFloat, 5, comms[7], streams[7]);
  ncclGroupEnd();
  ncclGroupStart();
  ncclRecv(d_buffers[5] + 6 * chunkSize, chunkSize, ncclFloat, 7, comms[5], streams[5]);
  ncclGroupEnd();
  ncclGroupEnd();

  hipSetDevice(devs[0]);
  hipEventRecord(stop, streams[0]);
  hipEventSynchronize(stop);
}

void stragglar_allreduce_delay(float** d_buffers, float** d_tempbufs, int* devs, hipStream_t* streams, ncclComm_t* comms, ncclComm_t* subComms, hipEvent_t start, hipEvent_t stop, int numRanks, size_t size, clock_t sleep_cycles) {
  size_t chunkSize = size / (numRanks - 1);
  hipSetDevice(devs[0]);
  hipEventRecord(start, streams[0]);

  hipSetDevice(devs[7]);
  gpu_sleep_kernel<<<1, 1, 0, streams[7]>>>(sleep_cycles);

  // Synchronize to make sure everything is idle
  for (int r = 0; r < numRanks - 1; ++r) {
    hipSetDevice(devs[r]);
    hipStreamSynchronize(streams[r]);
  }

  ncclGroupStart();
  for (int r = 0; r < numRanks - 1; ++r) {
    hipSetDevice(devs[r]);
    ncclReduceScatter(d_buffers[r], d_buffers[r] + (r * chunkSize), chunkSize, ncclFloat, ncclSum, subComms[r], streams[r]);
  }
  ncclGroupEnd();

  stragglar_allreduce_helper(d_buffers, d_tempbufs, devs, streams, comms, start, stop, numRanks, chunkSize);
}

void stragglar_allreduce(float** d_buffers, float** d_tempbufs, int* devs, hipStream_t* streams, ncclComm_t* comms, hipEvent_t start, hipEvent_t stop, int numRanks, size_t size) {
    size_t chunkSize = size / (numRanks - 1);
    hipSetDevice(devs[0]);
    hipEventRecord(start, streams[0]);
    stragglar_allreduce_helper(d_buffers, d_tempbufs, devs, streams, comms, start, stop, numRanks, chunkSize);
}

int main(int argc, char* argv[]) {
  const int numRanks = 8;

  int version;
  ncclGetVersion(&version);

  // printf("NCCL version %d\n", version);
  if (argc != 5) {
    fprintf(stderr, "Usage: %s <bufferSize> <algorithm> <numIters> <sleepTimeMs>\n", argv[0]);
    exit(EXIT_FAILURE);
  }

  size_t bytes = (size_t)strtoull(argv[1], NULL, 10);
  size_t size = bytes / sizeof(float);
  const char* alg = argv[2];
  int numIters = atoi(argv[3]);
  float sleepTime = atof(argv[4]);

  if (strcmp(alg, "ring") != 0 && strcmp(alg, "rhd") != 0 &&
      strcmp(alg, "direct") != 0 && strcmp(alg, "stragglar") != 0) {
    fprintf(stderr, "Invalid algorithm: %s\n", alg);
    exit(EXIT_FAILURE);
  }

  // Check GPUs
  int nGPUs = 0;
  CHECK_CUDA(hipGetDeviceCount(&nGPUs));
  if (nGPUs < numRanks) {
    printf("Need at least %d GPUs\n", numRanks);
    return -1;
  }

  int devs[numRanks];
  for (int i = 0; i < numRanks; ++i) devs[i] = i;

  // Allocate device buffers
  float* d_buffers[numRanks];
  float* d_tempbufs[numRanks];
  hipStream_t streams[numRanks];
  ncclComm_t comms[numRanks];
  ncclComm_t subComms[numRanks - 1];
 
  clock_t sleep_cycles;
  if (sleepTime >= 0) {
    sleep_cycles = calculate_sleep_cycles(sleepTime, devs);
    printf("Sleep cycles: %ld\n", sleep_cycles);
  }

  hipSetDevice(devs[0]);
  hipEvent_t start, stop;
  CHECK_CUDA(hipEventCreate(&start));
  CHECK_CUDA(hipEventCreate(&stop));

  CHECK_NCCL(ncclCommInitAll(comms, numRanks, devs));
  if (sleepTime >= 0) {
    CHECK_NCCL(ncclCommInitAll(subComms, numRanks - 1, NULL));
  }

  size_t chunkSize;

  if (strcmp(argv[2], "stragglar") == 0) {
    chunkSize = size / (numRanks - 1);
  }
  else if (strcmp(argv[2], "direct") == 0) {
    chunkSize = size;
  }
  else {
    chunkSize = size / numRanks;
  }

  for (int i = 0; i < numRanks; ++i) {
    CHECK_CUDA(hipSetDevice(devs[i]));
    CHECK_CUDA(hipStreamCreate(&streams[i]));
    CHECK_CUDA(hipMallocAsync(&d_buffers[i], size * sizeof(float), streams[i]));
    if (strcmp(argv[2], "ring") == 0 || strcmp(argv[2], "stragglar") == 0) {
      CHECK_CUDA(hipMallocAsync(&d_tempbufs[i], chunkSize * sizeof(float), streams[i]));
    }
    else if (strcmp(argv[2], "rhd") == 0) {
      CHECK_CUDA(hipMallocAsync(&d_tempbufs[i], 4 * chunkSize * sizeof(float), streams[i]));
    }
    else if (strcmp(argv[2], "direct") == 0) {
      CHECK_CUDA(hipMallocAsync(&d_tempbufs[i], size * sizeof(float), streams[i]));
    }
  }

  // warmup
  std::vector<std::vector<std::pair<int, int>>> steps = {{ {0,3}, {2,6}, {1,5}, {4,7} }};
  for (int iter = 0; iter < 10; ++iter) {
    for (const auto& step : steps) {
      ncclGroupStart();
      for (const auto& [src, dst] : step) {
        ncclSend(d_buffers[src], chunkSize, ncclFloat, dst, comms[src], streams[src]);
        ncclRecv(d_tempbufs[src], chunkSize, ncclFloat, dst, comms[src], streams[src]);
  
        ncclSend(d_buffers[dst], chunkSize, ncclFloat, src, comms[dst], streams[dst]);
        ncclRecv(d_tempbufs[dst], chunkSize, ncclFloat, src, comms[dst], streams[dst]);
      }
      ncclGroupEnd();
    }
  }

  printf("algorithm,buffer_size_bytes,iteration,delay,runtime_ms,BW(GB/s)\n");
  for (int iter = 0; iter < numIters + 1; ++iter) {
    // Reset buffers if needed (same init pattern as above)
    for (int i = 0; i < numRanks; ++i) {
      CHECK_CUDA(hipSetDevice(devs[i]));
      if (sleepTime < 0 && strcmp(alg,"direct")==0 && i < numRanks-1) {
        fill_pattern<<<(size+255)/256, 256, 0, streams[i] >>>(d_buffers[i], 28.f, size);
      } else{
        fill_pattern<<<(size+255)/256, 256, 0, streams[i] >>>(d_buffers[i], float(i+1), size);
      }
      if (sleepTime < 0 && strcmp(alg, "stragglar") == 0 && i < numRanks - 1) {
        fill_pattern<<< (chunkSize+255)/256, 256, 0, streams[i] >>>(d_buffers[i] + i*chunkSize, 28.f, chunkSize);
      }
    }

    for (int i = 0; i < numRanks; ++i) {
      CHECK_CUDA(hipSetDevice(devs[i]));
      CHECK_CUDA(hipStreamSynchronize(streams[i]));
    }

  // Run algorithm
    if (sleepTime >= 0) {
      if (strcmp(alg, "ring") == 0)
        ring_allreduce_delay(d_buffers, d_tempbufs, devs, streams, comms, start, stop, numRanks, size, sleep_cycles);
      else if (strcmp(alg, "rhd") == 0)
        rhd_allreduce_delay(d_buffers, d_tempbufs, devs, streams, comms, start, stop, numRanks, size, sleep_cycles);
      else if (strcmp(alg, "stragglar") == 0)
        stragglar_allreduce_delay(d_buffers, d_tempbufs, devs, streams, comms, subComms, start, stop, numRanks, size, sleep_cycles);
      else if (strcmp(alg, "direct") == 0)
        direct_allreduce_delay(d_buffers, d_tempbufs, devs, streams, comms, subComms, start, stop, numRanks, size, sleep_cycles);
    } else {
      if (strcmp(alg, "ring") == 0)
        ring_allreduce(d_buffers, d_tempbufs, devs, streams, comms, start, stop, numRanks, size);
      else if (strcmp(alg, "rhd") == 0) 
        rhd_allreduce(d_buffers, d_tempbufs, devs, streams, comms, start, stop, numRanks, size);
      else if (strcmp(alg, "stragglar") == 0) {
        stragglar_allreduce(d_buffers, d_tempbufs, devs, streams, comms, start, stop, numRanks, size);
      }
      else if (strcmp(alg, "direct") == 0)
        direct_allreduce(d_buffers, d_tempbufs, devs, streams, comms, start, stop, numRanks, size);
    }
    float ms;
    float bw;
    hipSetDevice(devs[0]);
    hipEventElapsedTime(&ms, start, stop);
    if (iter == 0) continue;
    if (sleepTime > 0) {
      bw = (float)size * sizeof(float) / 1024.0 / 1024.0 / 1024.0 * 1000.0 / (ms - sleepTime);
    }
    else {
      bw = (float)size * sizeof(float) / 1024.0 / 1024.0 / 1024.0 * 1000.0 / ms;
    }
    printf("%s,%zu,%d,%.3f,%.3f,%.3f\n",
      alg,
      (size_t)size * sizeof(float),   // bytes, still a size_t
      iter,
      sleepTime,
      ms,
      bw);
  }
  
  float* hostOut = (float*)malloc(size * sizeof(float));
  for (int r = 0; r < numRanks; ++r) {
    CHECK_CUDA(hipSetDevice(devs[r]));
    CHECK_CUDA(hipMemcpy(hostOut, d_buffers[r],
                          size * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < size; ++i) {
      assert(hostOut[i] == 36.0);
    }
  }
  free(hostOut);

  for (int i = 0; i < numRanks; ++i) {
    hipSetDevice(devs[i]);
    hipFree(d_buffers[i]);
    hipFree(d_tempbufs[i]);
    hipStreamDestroy(streams[i]);  // Streams are local to each device
    ncclCommDestroy(comms[i]);      // Safe last
    if (sleepTime >= 0 && i < numRanks - 1) {
      ncclCommDestroy(subComms[i]);
    }
    printf("Rank %d, done\n", i);
  }
  

  return 0;
}